#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include <time.h>
#include "CImg.h"

using namespace std;
using namespace cimg_library;

#include <iostream>
#define BLOCK_SIZE 32
#define WA 512   
#define HA 512     
#define HC 3     
#define WC 3
#define WB (WA - WC + 1)
#define HB (HA chanel- HC + 1)

typedef unsigned char chanel;
typedef unsigned int int64_t;
typedef float REAL;

using namespace std;
//Funcion kernel
__global__ void convolution(REAL* A, REAL* B, REAL* C, int numARows, int numACols, int numBRows, int numBCols, int numCRows, int numCCols)
{
	int col = blockIdx.x * (BLOCK_SIZE - WC + 1) + threadIdx.x;
	int row = blockIdx.y * (BLOCK_SIZE - WC + 1) + threadIdx.y;
	int row_i = row - WC + 1;
	int col_i = col - WC + 1;

	REAL tmp = 0;

	__shared__ REAL shm[BLOCK_SIZE][BLOCK_SIZE];//variable compartida

	if (row_i < WA && row_i >= 0 && col_i < WA && col_i >= 0)
	{
		shm[threadIdx.y][threadIdx.x] = A[col_i * WA + row_i];
	}
	else
	{
		shm[threadIdx.y][threadIdx.x] = 0;
	}

	__syncthreads();

	if (threadIdx.y < (BLOCK_SIZE - WC + 1) && threadIdx.x < (BLOCK_SIZE - WC + 1) && row < (WB - WC + 1) && col < (WB - WC + 1))
	{
		for (int i = 0; i< WC;i++)
			for (int j = 0;j<WC;j++)
				tmp += shm[threadIdx.y + i][threadIdx.x + j] * C[j*WC + i];
		B[col*WB + row] = tmp;
	}
}


int main(int argc, char** argv)
{
	CImg<chanel> src("image.jpg");
    int width = src.width();
    int height = src.height();
    
    unsigned long size = src.size();
    
    int A[width][width];
    
	hipError_t error;
	hipEvent_t start_G, stop_G;

	hipEventCreate(&start_G);
	hipEventCreate(&stop_G);

	int64_t size_A = WA * HA;
	int64_t mem_size_A = sizeof(REAL) * size_A;
	REAL* h_A = (REAL*)malloc(mem_size_A);

	int64_t size_B = WB * HB;
	int64_t mem_size_B = sizeof(REAL) * size_B;
	REAL* h_B = (REAL*)malloc(mem_size_B);

	int64_t size_C = WC * HC;
	int64_t mem_size_C = sizeof(REAL) * size_C;
	REAL* h_C = (REAL*)malloc(mem_size_C);


	
	/*Usando filtro de sobel*/
	h_C[0]=-1;h_C[1]=-2;h_C[2]=-1;
	h_C[3]=0;h_C[4]=0;h_C[5]=0;
	h_C[6]=1;h_C[7]=2;h_C[8]=1;
	

	int n=0;
	for (int x=0;x<width;x++){
		for (int y=0;y<width;y++){
			chanel r =(REAL)src(x,y,0,0);
			chanel g =(REAL)src(x,y,0,1);
			chanel b =(REAL)src(x,y,0,2);
									
			int _gray = (int64_t)((REAL)(r + g + b) / 3.0f + 0.5);
			
			src(x,y,0,0)=_gray;
			src(x,y,0,1)=_gray;
			src(x,y,0,2)=_gray;
			
			A[x][y] =_gray;
			h_A[n]=_gray;
			n++;
		}	

	}
	
	src.display();
	REAL* d_A;
	REAL* d_B;
	REAL* d_C;

	hipMalloc((void**)&d_A, mem_size_A);
	hipMalloc((void**)&d_B, mem_size_B);
	hipMalloc((void**)&d_C, mem_size_C);

	hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, mem_size_C, hipMemcpyHostToDevice);

	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((WB - 1) / (BLOCK_SIZE - WC + 1), (WB - 1) / (BLOCK_SIZE - WC + 1));

	convolution << < grid, threads >> >(d_A, d_B, d_C, HA, WA, HB, WB, HC, WC);
	
	hipMemcpy(h_B, d_B, mem_size_B, hipMemcpyDeviceToHost);

	for (int i = 0;i < HB;i++)
	{
		for (int j = 0;j < WB;j++)
		{
			src(i,j,0,0)=h_B[i*HB + j];
			src(i,j,0,1)=h_B[i*HB + j];
			src(i,j,0,2)=h_B[i*HB + j];
			
		}
	}
	

	free(h_A);
	free(h_B);
	free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	src.display();
	return 0;
}
